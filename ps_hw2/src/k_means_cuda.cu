#include <iostream>

#include "k_means_cuda.h"
#include "common.h"

#include <hip/hip_runtime.h>
#include "ext/hip/hip_runtime_api.h"

int k_means_cuda(int n_points, real *points, struct options_t *opts,
  int* point_cluster_ids, real** centroids) {

  int device_id = gpuGetMaxGflopsDeviceId();

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, device_id));
  checkCudaErrors(hipSetDevice(device_id));
  DEBUG_PRINT(printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", device_id, deviceProp.name));

  bool done = false;
  int iterations = 0;
  int k = opts->n_clusters;
  int d = opts->dimensions;

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  real *d_points;
  checkCudaErrors(hipMalloc(&d_points, n_points * d * sizeof(real)));

  real *old_centroids;
  checkCudaErrors(hipMalloc(&old_centroids, k * d * sizeof(real)));

  real *new_centroids;
  checkCudaErrors(hipMalloc(&new_centroids, k * d * sizeof(real)));

  real *point_centroid_distances;
  checkCudaErrors(hipMalloc(&point_centroid_distances, n_points * k * sizeof(real)));

  int *d_pointcluster_ids;
  checkCudaErrors(hipMalloc(&point_centroid_distances, n_points * sizeof(int)));
  // dv_int unsorted_d_point_cluster_ids(n_points);

  int *d_k_counts;
  checkCudaErrors(hipMalloc(&point_centroid_distances, k * sizeof(int)));

  // copy data
  // timer code - 0_Simple/simpleMultiCopy/simpleMultiCopy.cu
  hipEventRecord(start, 0);
  checkCudaErrors(hipMemcpyAsync(d_points, points, n_points * d * sizeof(real),
                                  hipMemcpyHostToDevice, 0));

  checkCudaErrors(hipMemcpyAsync(old_centroids, *centroids, k * d * sizeof(real),
                                  hipMemcpyHostToDevice, 0));
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  float memcpy_h2d_time;
  hipEventElapsedTime(&memcpy_h2d_time, start, stop);
  DEBUG_PRINT(printf("Device to host: %f ms \n", memcpy_h2d_time));


  // while(!done) {
  //   DEBUG_OUT("old centroid");
  //   D_PRINT_ALL(old_centroids);

  //   distance_component distance_component(d, k,
  //       raw_pointer_cast(d_points.data()), raw_pointer_cast(old_centroids.data()));

  //   // compute the distances for each point/centroid
  //   // https://github.com/NVIDIA/thrust/blob/1d067bdba7aaca4b53cd4d43b98ac180a0308446/examples/sum_rows.cu
  //   reduce_by_key(
  //     make_transform_iterator(counting_iterator<int>(0), _1 / d),
  //     make_transform_iterator(counting_iterator<int>(n_points * k * d), _1 / d),
  //     make_transform_iterator(counting_iterator<int>(0),  distance_component),
  //     make_discard_iterator(), //discard keys
  //     point_centroid_distances.begin()
  //   );

  //   DEBUG_OUT("centroid distance");
  //   D_PRINT_ALL(point_centroid_distances);

  //   // reduce to the index of the nearest centroid
  //   reduce_by_key(
  //     make_transform_iterator(counting_iterator<int>(0), _1 / k),
  //     make_transform_iterator(counting_iterator<int>(n_points * k), _1 / k),
  //     make_zip_iterator(
  //       make_tuple(
  //         point_centroid_distances.begin(),
  //         counting_iterator<int>(0))),
  //     make_discard_iterator(),
  //     make_zip_iterator(
  //       make_tuple(
  //         make_discard_iterator(), //discard distance values
  //         unsorted_d_point_cluster_ids.begin())),
  //     equal_to<int>(),
  //     maximum_by_first()
  //   );

  //   transform(
  //     unsorted_d_point_cluster_ids.begin(),
  //     unsorted_d_point_cluster_ids.end(),
  //     unsorted_d_point_cluster_ids.begin(),
  //     _1 % k
  //   );

  //   DEBUG_OUT("%k unsorted_d_point_cluster_ids");
  //   D_PRINT_ALL(unsorted_d_point_cluster_ids);

  //   // zero the new centroids
  //   fill(new_centroids.begin(), new_centroids.end(), 0);

  //   // point_component_centroid_id point_component_centroid_id(d,
  //   //     raw_pointer_cast(d_point_cluster_ids.data()),
  //   //     raw_pointer_cast(new_centroids.data()));
  //   // dv_int v(k*d);

  //   //compute the sum of points for each centroid
  //   // reduce_by_key(
  //   //   make_transform_iterator(counting_iterator<int>(0), point_component_centroid_id),
  //   //   make_transform_iterator(counting_iterator<int>(n_points * d), point_component_centroid_id),
  //   //   d_points.begin(),
  //   //   v.begin(),
  //   //   // make_discard_iterator(), //discard keys
  //   //   new_centroids.begin()
  //   // );
  //   // ok, no fancy reduce_by_key

  //   centroid_means centroid_means(d,
  //     raw_pointer_cast(unsorted_d_point_cluster_ids.data()),
  //     raw_pointer_cast(new_centroids.data()));

  //   for_each(
  //     make_zip_iterator(
  //       make_tuple(d_points.begin(), counting_iterator<int>(0))),
  //     make_zip_iterator(
  //       make_tuple(d_points.end(), counting_iterator<int>(n_points * d))),
  //     centroid_means
  //   );

  //   DEBUG_OUT("new_centroids sums");
  //   D_PRINT_ALL(new_centroids);
  //   // D_PRINT_ALL(v);

  //   d_point_cluster_ids = unsorted_d_point_cluster_ids;

  //   sort(
  //     d_point_cluster_ids.begin(),
  //     d_point_cluster_ids.end()
  //   );

  //   //compute the point count for each centroid
  //   auto new_end = reduce_by_key(
  //     d_point_cluster_ids.begin(),
  //     d_point_cluster_ids.end(),
  //     make_constant_iterator(1),
  //     d_k_count_keys.begin(),
  //     d_k_counts.begin()
  //   );

  //   // zero out any stragglers
  //   fill(new_end.second, d_k_counts.end(), 0);

  //   DEBUG_OUT("k_counts");
  //   D_PRINT_ALL(d_k_counts);
  //   D_PRINT_ALL(d_k_count_keys);
  //   // assert(new_end.first == d_k_count_keys.end());

  //   for (int i = 0; i < d; i++) {
  //     for_each(
  //       make_zip_iterator(
  //         make_tuple(d_k_count_keys.begin(), d_k_counts.begin())),
  //       make_zip_iterator(
  //         make_tuple(new_end.first, new_end.second)),
  //       centroid_divide(d, i, raw_pointer_cast(new_centroids.data()))
  //     );
  //   }

  //   DEBUG_OUT("new_centroids means");
  //   D_PRINT_ALL(new_centroids);

  //   // swap centroids
  //   swap(new_centroids, old_centroids);

  //   real l1_thresh = opts->threshold/d;

  //   bool converged = transform_reduce(
  //     make_zip_iterator(
  //       make_tuple(new_centroids.begin(), old_centroids.begin())),
  //     make_zip_iterator(
  //       make_tuple(new_centroids.end(), old_centroids.end())),
  //     l1_op(l1_thresh),
  //     true,
  //     logical_and<bool>()
  //   );

  //   iterations++;
  //   done = (iterations > opts->max_iterations) || converged;
  // }
  // // release the other centroids buffer

  // DEBUG_OUT(iterations > opts->max_iterations ? "Max iterations reached!" : "Converged!" );

  // copy(old_centroids.begin(), old_centroids.end(), *centroids);
  // copy(unsorted_d_point_cluster_ids.begin(), unsorted_d_point_cluster_ids.end(), point_cluster_ids);

  // return iterations;
  return 0;
}
