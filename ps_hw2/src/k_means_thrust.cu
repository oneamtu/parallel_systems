#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/copy.h>

#include "k_means_thrust.h"
#include "common.h"

typedef thrust::device_vector<real> dv_real;
typedef thrust::device_vector<int> dv_int;
typedef thrust::host_vector<real> hv_real;

__device__ double datomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
        __double_as_longlong(val +
          __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

#ifdef DEBUG
#define D_PRINT_POINT(x, d, i) { \
thrust::copy_n( \
    x.begin() + i*d, \
    d, \
    std::ostream_iterator<real>(std::cerr, ", ") \
  ); \
  std::cerr << std::endl;\
}
#define D_PRINT_ALL(x) { \
thrust::copy( \
    x.begin(), \
    x.end(), \
    std::ostream_iterator<real>(std::cerr, ", ") \
  ); \
  std::cerr << std::endl;\
}
#else
#define D_PRINT_POINT(x, d, i)
#define D_PRINT_ALL(x)
#endif

typedef thrust::tuple<real, int> real_indexed;

// map an index -> component l of point index i and centroid index j
// and then compute the distance component of l
struct distance_component : public thrust::unary_function<int, real> {
  const int d, k;
  const real *points, *centroids;

  distance_component(int _d, int _k, real *_points, real *_centroids)
    : d(_d), k(_k), points(_points), centroids(_centroids) {}

  __host__ __device__
  real operator()(int index) {
    int i = (index / d) / k;
    int j = (index / d) % k;
    int l = index % d;

    // TODO:
    return POW2(points[i*d+l] - centroids[j*d+l]);
    // return POW2(points[i*d+l]) - 2*points[i*d+l]*centroids[j*d+l] + POW2(centroids[j*d+l]);
  }
};

// sum up each point component into a new centroid
struct centroid_means : public thrust::unary_function<void, real_indexed> {
  const int d;
  const int *d_point_cluster_ids, *d_k_counts;
  real *centroids;

  centroid_means(int _d, int *_d_point_cluster_ids, int *_d_k_counts, real *_centroids)
    : d(_d), d_point_cluster_ids(_d_point_cluster_ids),
    d_k_counts(_d_k_counts), centroids(_centroids) {}

  __device__
  void operator()(real_indexed real_index) {
    real value = thrust::get<0>(real_index);
    int index = thrust::get<1>(real_index);
    int target_centroid_id = d_point_cluster_ids[index / d];
    int target_centroid_component_id = target_centroid_id*d + index % d;

    datomicAdd(centroids + target_centroid_component_id, value / d_k_counts[target_centroid_id]);
  }
};

//functor that makes point component indexes equivalent based on a map
//in a n_points * k array
struct point_component_centroid_id : public thrust::binary_function<int, int, int> {
  const int d;
  const int *point_cluster_ids;
  const real *centroids;

  point_component_centroid_id(int _d, int *_point_cluster_ids, real *_centroids)
    : d(_d), point_cluster_ids(_point_cluster_ids), centroids(_centroids) {}

  __host__ __device__
  int operator()(int index) {
    return point_cluster_ids[index / d] * d + index % d;
  }
};

//binary function for a maximum on the first member of a tuple
struct maximum_by_first : public thrust::binary_function<real_indexed, real_indexed, real_indexed> {
  maximum_by_first() {}

  __host__ __device__
  real_indexed operator()(real_indexed x_1, real_indexed x_2) {
    return thrust::get<0>(x_1) < thrust::get<0>(x_2) ? x_1 : x_2;
  }
};

int k_means_thrust(int n_points, real *points, struct options_t *opts,
  int* point_cluster_ids, real** centroids) {

  using namespace thrust;

  //https://github.com/NVIDIA/thrust/blob/1d067bdba7aaca4b53cd4d43b98ac180a0308446/examples/lambda.cu
  using namespace thrust::placeholders;

  bool done = false;
  int iterations = 0;
  int k = opts->n_clusters;
  int d = opts->dimensions;

  dv_real d_points(points, points + n_points * d);

  dv_real old_centroids(*centroids, *centroids + k * d);
  dv_real new_centroids(k * d);

  dv_real point_centroid_distances(n_points * k);
  dv_int d_point_cluster_ids(n_points);

  dv_int d_k_counts(k);
  dv_int d_k_count_keys(k);

  while(!done) {
    DEBUG("old centroid");
    D_PRINT_ALL(old_centroids);

    distance_component distance_component(d, k,
        raw_pointer_cast(d_points.data()), raw_pointer_cast(old_centroids.data()));

    // compute the distances for each point/centroid
    // https://github.com/NVIDIA/thrust/blob/1d067bdba7aaca4b53cd4d43b98ac180a0308446/examples/sum_rows.cu
    reduce_by_key(
      make_transform_iterator(counting_iterator<int>(0), _1 / d),
      make_transform_iterator(counting_iterator<int>(n_points * k * d), _1 / d),
      make_transform_iterator(counting_iterator<int>(0),  distance_component),
      make_discard_iterator(), //discard keys
      point_centroid_distances.begin()
    );

    DEBUG("centroid distance");
    D_PRINT_ALL(point_centroid_distances);

    // reduce to the index of the nearest centroid
    reduce_by_key(
      make_transform_iterator(counting_iterator<int>(0), _1 / k),
      make_transform_iterator(counting_iterator<int>(n_points * k), _1 / k),
      make_zip_iterator(
        make_tuple(
          point_centroid_distances.begin(),
          counting_iterator<int>(0))),
      make_discard_iterator(),
      make_zip_iterator(
        make_tuple(
          make_discard_iterator(), //discard distance values
          d_point_cluster_ids.begin())),
      equal_to<int>(),
      maximum_by_first()
    );

    transform(
      d_point_cluster_ids.begin(),
      d_point_cluster_ids.end(),
      d_point_cluster_ids.begin(),
      _1 % k
    );

    DEBUG("%k d_point_cluster_ids");
    D_PRINT_ALL(d_point_cluster_ids);

    sort(
      d_point_cluster_ids.begin(),
      d_point_cluster_ids.end()
    );

    //compute the point count for each centroid
    auto new_end = reduce_by_key(
      d_point_cluster_ids.begin(),
      d_point_cluster_ids.end(),
      make_constant_iterator(1),
      d_k_count_keys.begin(),
      d_k_counts.begin()
    );

    sort_by_key(
      d_k_count_keys.begin(),
      new_end.first,
      d_k_counts.begin()
    );
    // zero out any stragglers
    fill(new_end.second, d_k_counts.end(), 0);

    DEBUG("k_counts");
    D_PRINT_ALL(d_k_counts);
    D_PRINT_ALL(d_k_count_keys);
    DEBUG(new_end.first - d_k_count_keys.end());

    // zero the new centroids
    fill(new_centroids.begin(), new_centroids.end(), 0);

    // point_component_centroid_id point_component_centroid_id(d,
    //     raw_pointer_cast(d_point_cluster_ids.data()),
    //     raw_pointer_cast(new_centroids.data()));
    // dv_int v(k*d);

    //compute the sum of points for each centroid
    // reduce_by_key(
    //   make_transform_iterator(counting_iterator<int>(0), point_component_centroid_id),
    //   make_transform_iterator(counting_iterator<int>(n_points * d), point_component_centroid_id),
    //   d_points.begin(),
    //   v.begin(),
    //   // make_discard_iterator(), //discard keys
    //   new_centroids.begin()
    // );
    // ok, no fancy reduce_by_key

    centroid_means centroid_means(d,
      raw_pointer_cast(d_point_cluster_ids.data()),
      raw_pointer_cast(d_k_counts.data()),
      raw_pointer_cast(new_centroids.data()));

    for_each(
      make_zip_iterator(
        make_tuple(d_points.begin(), counting_iterator<int>(0))),
      make_zip_iterator(
        make_tuple(d_points.end(), counting_iterator<int>(n_points * d))),
      centroid_means
    );

    DEBUG("new_centroids sums");
    D_PRINT_ALL(new_centroids);
    // D_PRINT_ALL(v);

    // swap centroids
    swap(new_centroids, old_centroids);

    iterations++;
    done = (iterations > opts->max_iterations);
    //   converged(k, d, opts->threshold, centroids_1, centroids_2); TODO
  }
  // release the other centroids buffer

  DEBUG(iterations > opts->max_iterations ? "Max iterations reached!" : "Converged!" );

  copy(old_centroids.begin(), old_centroids.end(), *centroids);
  copy(d_point_cluster_ids.begin(), d_point_cluster_ids.end(), point_cluster_ids);

  return iterations;
}
